// a. Write a kernel that has each thread produce one output matrix row. 
// Fill in the execution configuration parameters for the design.
// Kernel: Each thread computes one row of output_matrix
// output_matrix = input_matrix1 × input_matrix2, where input_matrix1 is total_rows×K, input_matrix2 is K×total_cols, output_matrix is total_rows×total_cols
__global__ void matMulRow(float *input_matrix1, float *input_matrix2, float *output_matrix, int total_rows, int K, int total_cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (row < total_rows) {
        // Compute entire row
        for (int col = 0; col < total_cols; col++) {
            float sum = 0.0f;
            for (int k = 0; k < K; k++) {
                sum += input_matrix1[row * K + k] * input_matrix2[k * total_cols + col];
            }
            output_matrix[row * total_cols + col] = sum;
        }
    }
}

#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    // Matrix dimensions
    int total_rows = 512;
    int K = 256;
    int total_cols = 512;
    
    size_t size1 = total_rows * K * sizeof(float);
    size_t size2 = K * total_cols * sizeof(float);
    size_t size_out = total_rows * total_cols * sizeof(float);
    
    // Allocate host memory
    float *h_input1 = (float*)malloc(size1);
    float *h_input2 = (float*)malloc(size2);
    float *h_output = (float*)malloc(size_out);
    
    // Initialize input matrices
    for (int i = 0; i < total_rows * K; i++) h_input1[i] = 1.0f;
    for (int i = 0; i < K * total_cols; i++) h_input2[i] = 1.0f;
    
    // Allocate device memory
    float *d_input1, *d_input2, *d_output;
    hipMalloc(&d_input1, size1);
    hipMalloc(&d_input2, size2);
    hipMalloc(&d_output, size_out);
    
    // Copy to device
    hipMemcpy(d_input1, h_input1, size1, hipMemcpyHostToDevice);
    hipMemcpy(d_input2, h_input2, size2, hipMemcpyHostToDevice);
    
    // Launch kernel
    int threadsPerBlock = 256;
    int numBlocks = (total_rows + threadsPerBlock - 1) / threadsPerBlock;
    matMulRow<<<numBlocks, threadsPerBlock>>>(d_input1, d_input2, d_output, total_rows, K, total_cols);
    
    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch error: %s\n", hipGetErrorString(err));
        return 1;
    }
    
    // Wait for kernel to complete
    hipDeviceSynchronize();
    
    // Copy result back
    hipMemcpy(h_output, d_output, size_out, hipMemcpyDeviceToHost);
    
    // Verify result (first element should be K * 1.0 = 256.0)
    printf("Result[0][0] = %.2f (expected %.2f)\n", h_output[0], (float)K);
    
    // Cleanup
    hipFree(d_input1);
    hipFree(d_input2);
    hipFree(d_output);
    free(h_input1);
    free(h_input2);
    free(h_output);
    
    return 0;
}
